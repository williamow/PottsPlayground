#include "hip/hip_runtime.h"
#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "NumCuda.hpp"
#include "PieceWiseLinear.cpp"


/*
cuda kernel for sampling a single replicate of a Potts model.

weights: nNHPPs x nNHPPs
	matrix of weights in the boltzmann machine.  Symmetric, row/column ordering can be chosen to maximize performance.
partitions: nNHPPs
    indicates which Potts group each NHPP belongs to; maps NHPP index to addresses in the state vectors.
    Each integer value is therefore in the range [0, nPartitions)
states: nReplicates x nPartitions
	Each state value indicates the active state of a Potts node, represented by the integer index of the NHPP,
    so that the values in the state vectors directly index to which values in the weight matrix are 'active' for the given state.
    Two sets of state memory are provided - one for keeping track of the best-so-far state,
    and another for holding the working, evolving state that the algorithm acts upon. 
*/

__host__ __device__ void PottsFullWeightsHost(
    NumCuda<float> &weights, //if these are not passed by reference, the pointers within cannot be accesssed... I don't know why
    NumCuda<int> &partitions,
    NumCuda<int> &working_states,
    NumCuda<int> &best_states,
    NumCuda<float> &working_energies,
    NumCuda<float> &best_energies,
    int minIter,
    int maxIter,
    PieceWiseLinear PwlTemp,
    float e_th, //energy threshold to trigger premature termination
    volatile int *global_halt //bit that tells all processes to exit, set by a single process upon finding a 'solution'
    )
{
    
    //index for which replicate this thread is computing; only matters for writing the final best state back to global memory
    #ifdef __CUDA_ARCH__
        int replicate_index = blockDim.x * blockIdx.x + threadIdx.x;
    #else
        int replicate_index = 0;
    #endif
    
    //get some dimensional values from the passed NumCuda arrays:
    int nReplicates = working_energies.dims[0];
    int nNHPPs = weights.dims[0];
    int nPartitions = working_states.dims[1];
    if (replicate_index + 1 > nReplicates) return;

    //create local references to the state vectors used in this particular thread:
    int *MiWrk = &working_states(replicate_index, 0);
    int *MiBest = &best_states(replicate_index, 0);

    //rng initialization.  Different libraries are used, depending on if we're compiling for host or gpu
    #ifdef __CUDA_ARCH__
        hiprandState RngState;
        hiprand_init(replicate_index, minIter, 0, &RngState); //use thread index number and starting iteration to seed the rng
        #define RngUniform() hiprand_uniform(&RngState)
        #define RngInteger() hiprand(&RngState)
        // printf("running in Cuda kernel\n");

    #else
        std::mt19937 RngGenerator(replicate_index + minIter*1000);
        std::uniform_real_distribution<float> uniform_distribution(0.0,1.0);
        #define RngUniform() uniform_distribution(RngGenerator)
        #define RngInteger() RngGenerator()

    #endif
    
    //total energy tracker initialization ===============================================

    if (minIter == 0){
        //then states should be newly initialized.
        //effectively initializes each Potts state to be the highest possible index for that state:
        for (int NHPP = 0; NHPP < nNHPPs; NHPP++){
            MiWrk[partitions(NHPP)] = NHPP;
            MiBest[partitions(NHPP)] = NHPP;}
    }
    else{
        //since a non-standard state format is used,
        //we need to convert the standard state (which is stored between runs)
        //to the internally-used non-standard state
        int last_partition = -1;
        for (int NHPP = 0; NHPP < nNHPPs; NHPP++){
            if (partitions(NHPP) > last_partition){
                last_partition = partitions(NHPP);
                MiWrk[last_partition] += NHPP;
                MiBest[last_partition] += NHPP;
            }
        }
    }
    
    // return;

    //calculate starting total energy:
    float current_e = 0;
    float lowest_e = 0;
    for (int i = 0; i < nPartitions; i++){
        for (int j = 0; j < nPartitions; j++){
            current_e += weights(MiWrk[i], MiWrk[j]);
            lowest_e += weights(MiBest[i], MiBest[j]);
            //weights contribute to the active energy when both sides of the weights are selected.
        }
    }
    current_e = current_e / 2;
    lowest_e = lowest_e / 2;

    if (lowest_e < e_th){
        *global_halt = replicate_index+1;
    }
            

    //main loop ==================================================================================
    for (int iter = minIter; iter < maxIter; iter++){
        if (*global_halt > 0)
            break; //if one of the threads has found a solution, all threads exit here

        int Mi_proposed = RngInteger()%nNHPPs;
        int Mi_current = MiWrk[partitions(Mi_proposed)];

        float dE = 0;
        //compute how much the energy would change.
        //assumes no weights between Mi_proposed and Mi_current, otherwise this calculation would be incorrect.
        for (int i = 0; i < nPartitions; i++){
            dE = dE + weights(Mi_proposed, MiWrk[i]) - weights(Mi_current, MiWrk[i]);
        }

        float T = PwlTemp.interp(iter);
        float flip_prob = exp(-dE/T);

        if (flip_prob >= RngUniform()){
            //update the state:
            MiWrk[partitions(Mi_proposed)] = Mi_proposed;
            //update the energy:
            current_e += dE;
            //and possibly update the lowest energy value:
            if (current_e < lowest_e){
                lowest_e = current_e;
                for (int i = 0; i < nPartitions; i++){
                    MiBest[i] = MiWrk[i];
                }
                if (lowest_e < e_th){
                    *global_halt = replicate_index+1;
                }
            }
        }
    }
    working_energies(replicate_index) = current_e;
    best_energies(replicate_index) = lowest_e;

    //convert internal state format back to general state format:
    int last_partition = -1;
    for (int NHPP = 0; NHPP < nNHPPs; NHPP++){
        if (partitions(NHPP) > last_partition){
            last_partition = partitions(NHPP);
            MiWrk[last_partition] -= NHPP;
            MiBest[last_partition] -= NHPP;
        }
    }
}



// ============================================================wrapper to call main code as a GPU kernel
__global__ void PottsFullWeightsDevice(
    NumCuda<float> weights,
    NumCuda<int> partitions,
    NumCuda<int> working_states,
    NumCuda<int> best_states,
    NumCuda<float> working_energies,
    NumCuda<float> best_energies,
    int minIter,
    int maxIter,
    PieceWiseLinear PwlTemp,
    float e_th, //energy threshold to trigger premature termination
    volatile int *global_halt //bit that tells all processes to exit, set by a single process upon finding a 'solution'
    )
{
    // return;
    PottsFullWeightsHost(
        weights, partitions,
        working_states, best_states,
        working_energies, best_energies,
        minIter, maxIter,
        PwlTemp, e_th, global_halt
    );
}